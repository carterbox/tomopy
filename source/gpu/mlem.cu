#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

#include "backend/cuda.hh"
#include "backend/cuda_algorithms.hh"
#include "backend/ranges.hh"
#include "common.hh"
#include "constants.hh"
#include "data.hh"
#include "utils.hh"

#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <memory>
#include <numeric>

//======================================================================================//

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_total;
extern nvtxEventAttributes_t nvtx_iteration;
extern nvtxEventAttributes_t nvtx_slice;
extern nvtxEventAttributes_t nvtx_projection;
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

namespace
{
GLOBAL_CALLABLE void
pixels_kernel(int p, int dy, int dx, int nx, int ny, float* glob_recon, const float* data)
{
    // dt == number of angles
    // dy == number of slices
    // dx == number of pixels
    // nx == ngridx
    // ny == ngridy

    extern __shared__ float sum[];  // size dy * nx

    // set shared to zero
    {
        auto range = grid_strided_range<device::gpu, 0>(dx);
        for(int i = range.begin(); i < range.end(); i += range.stride())
        {
            sum[i] = 0.0f;
        }
    }
    __syncthreads();

    for(int k = 0; k < dy; ++k)
    {
        float* _sum   = sum + k * dy;
        float* _recon = glob_recon + k * nx * ny;
        auto   range  = grid_strided_range<device::gpu, 0>(dx * nx);
        for(int i = range.begin(); i < range.end(); i += range.stride())
        {
            atomicAdd(&_sum[i % dx], _recon[i]);
        }
    }
    __syncthreads();

    for(int k = 0; k < dy; ++k)
    {
        const float* _data  = data + p * dx;
        float*       _sum   = sum + k * dy;
        float*       _recon = glob_recon + k * nx * ny;
        auto         range  = grid_strided_range<device::gpu, 0>(dx * nx);
        for(int i = range.begin(); i < range.end(); i += range.stride())
        {
            _recon[i] += _data[i] / _sum[i % dx];
        }
    }
}

//======================================================================================//

GLOBAL_CALLABLE void
update_kernel(float* recon, const float* update, const uint32_t* sum_dist, int dx,
              int size)
{
    if(dx == 0)
        return;
    auto  range = grid_strided_range<device::gpu, 0>(size);
    float fdx   = static_cast<float>(dx);
    for(int i = range.begin(); i < range.end(); i += range.stride())
    {
        uint32_t sum = sum_dist[i];
        float    upd = update[i];
        if(sum != 0 && update[i] == update[i])
            recon[i] *= upd / static_cast<float>(sum) / fdx;
    }
}

} // namespace

//======================================================================================//

void
mlem_cuda(const float* cpu_data, int dy, int dt, int dx, const float*, const float* theta,
          float* cpu_recon, int nx, int ny, int num_iter, RuntimeOptions* opts)
{
    printf("[%lu]> %s : nitr = %i, dy = %i, dt = %i, dx = %i, nx = %i, ny = %i\n",
           this_thread_id(), __FUNCTION__, num_iter, dy, dt, dx, nx, ny);

    // thread counter for device assignment
    static std::atomic<int> ntid;

    // compute some properties (expected python threads, max threads, device assignment)
    int pythread_num = ntid++;
    int device       = pythread_num % cuda::device_count();  // assign to device

    TIMEMORY_AUTO_TIMER("");

    // GPU allocated copies
    cuda::set_device(device);
    printf("[%lu] Running on device %i...\n", this_thread_id(), device);

    auto      interp       = opts->interpolation;
    uintmax_t recon_pixels = static_cast<uintmax_t>(dy * nx * ny);
    auto      params       = cuda::kernel_params(opts->block_size[0], opts->grid_size[0]);
    auto      streams      = cuda::stream_create(opts->pool_size);
    float*    update       = cuda::malloc<float>(recon_pixels);
    float*    plus_rot     = cuda::malloc<float>(dt * recon_pixels);
    float*    back_rot     = cuda::malloc<float>(dt * recon_pixels);
    float*    recon        = cuda::malloc<float>(recon_pixels);
    float*    data         = cuda::malloc<float>(dy * dt * dx);
    cuda::memcpy(recon, cpu_recon, recon_pixels, cuda::host_to_device_v, 0);
    cuda::memcpy(data, cpu_data, dy * dt * dx, cuda::host_to_device_v, 0);
    uint32_t* sum_dist = cuda::compute_sum_dist(dy, dt, dx, nx, ny, theta, params);

    TOMOPY_NVXT_RANGE_PUSH(&nvtx_total);

    auto get_stream = [&](const size_t& offset) {
        return streams.at(offset % streams.size());
    };

    auto sync_stream = [&](const size_t& beg, const size_t& end) {
        for(size_t i = beg; i < end; ++i)
            cuda::stream_sync(get_stream(i));
    };

    auto get_proj  = [&](float* arr, const int& p) { return arr + p * recon_pixels; };
    auto get_slice = [&](float* arr, const int& s) { return arr + s * nx * ny; };

    for(int i = 0; i < num_iter; i++)
    {
        // timing and profiling
        TIMEMORY_AUTO_TIMER("");
        TOMOPY_NVXT_RANGE_PUSH(&nvtx_iteration);
        START_TIMER(t_start);

        // reset global update and sum_dist
        cuda::memset(update, 0, recon_pixels, get_stream(0));
        cuda::memset(plus_rot, 0, dt * recon_pixels, get_stream(1));
        cuda::memset(back_rot, 0, dt * recon_pixels, get_stream(2));

        // sync
        sync_stream(0, 3);

        // execute the loop over slices and projection angles
        // loop over independent projection angles
        for(int p = 0; p < dt; ++p)
        {
            cuda::stream_t stream  = get_stream(p);
            float          theta_p = fmodf(theta[p], twopi);

            float* p_plus_rot = get_proj(plus_rot, p);
            for(int s = 0; s < dy; ++s)
            {
                const float* s_recon     = recon + s * nx * ny;
                float*       sp_plus_rot = p_plus_rot + s * nx * ny;
                cuda::rotate(sp_plus_rot, s_recon, -theta_p, nx, ny, stream, interp);
            }

            {
                int smem  = dy * dx * sizeof(float);
                int block = params.block;
                int grid  = params.compute(dx * nx);
                pixels_kernel<<<grid, block, smem, stream>>>(p, dy, dx, nx, ny, plus_rot,
                                                             data);
            }

            // calculate offset for the streams
            auto* p_back_rot = get_proj(back_rot, p);
            for(int s = 0; s < dy; ++s)
            {
                float* sp_plus_rot = get_slice(p_plus_rot, s);
                float* sp_back_rot = get_slice(p_back_rot, s);
                cuda::rotate(sp_back_rot, sp_plus_rot, theta_p, nx, ny, stream, interp);
            }

            CUDA_CHECK_LAST_ERROR(stream);
        }

        // update array
        for(int p = 0; p < dt; ++p)
        {
            cuda::stream_t stream = get_stream(p);
            int            block  = params.block;
            int            grid   = params.compute(recon_pixels);
            cuda::atomic_sum<<<grid, block, 0, stream>>>(update, get_proj(back_rot, p),
                                                         recon_pixels);
        }

        // update the global recon with global update and sum_dist
        int block = params.block;
        int grid  = params.compute(recon_pixels);
        update_kernel<<<grid, block>>>(recon, update, sum_dist, dx, recon_pixels);

        // stop profile range and report timing
        TOMOPY_NVXT_RANGE_POP(0);
        REPORT_TIMER(t_start, "iteration", i, num_iter);
    }

    for(auto& itr : streams)
        cuda::stream_sync(itr);
    cuda::stream_sync(0);

    // copy to cpu
    cuda::memcpy(cpu_recon, recon, recon_pixels, cuda::device_to_host_v, 0);

    // sync and destroy main stream
    cuda::stream_destroy(streams);

    // cleanup
    cuda::free(recon);
    cuda::free(data);
    cuda::free(update);
    cuda::free(sum_dist);

    TOMOPY_NVXT_RANGE_POP(0);

    // sync the device
    cuda::device_sync();
}

//======================================================================================//
