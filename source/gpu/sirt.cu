#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

#include "backend/cuda.hh"
#include "backend/cuda_algorithms.hh"
#include "backend/ranges.hh"
#include "common.hh"
#include "constants.hh"
#include "data.hh"
#include "utils.hh"

#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <memory>
#include <numeric>

//======================================================================================//

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_total;
extern nvtxEventAttributes_t nvtx_iteration;
extern nvtxEventAttributes_t nvtx_slice;
extern nvtxEventAttributes_t nvtx_projection;
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

namespace
{
GLOBAL_CALLABLE void
pixels_kernel(int p, int nx, int dx, float* recon, const float* data)
{
    auto range = grid_strided_range<device::gpu, 0>(dx);
    for(int d = range.begin(); d < range.end(); d += range.stride())
    {
        float sum = 0.0f;
        for(int i = 0; i < nx; ++i)
            sum += recon[i * nx + d];
        float upd = data[p * dx + d] - sum;
        for(int i = 0; i < nx; ++i)
            recon[i * nx + d] += upd;
    }
}

//======================================================================================//

GLOBAL_CALLABLE void
update_kernel(float* recon, const float* update, const uint32_t* sum_dist, int dx,
              int size)
{
    if(dx == 0)
        return;
    auto  range = grid_strided_range<device::gpu, 0>(size);
    float fdx   = static_cast<float>(dx);
    for(int i = range.begin(); i < range.end(); i += range.stride())
    {
        uint32_t sum = sum_dist[i];
        float    upd = update[i];
        if(sum != 0 && update[i] == update[i])
            recon[i] += upd / static_cast<float>(sum) / fdx;
    }
}

//======================================================================================//

void
compute_projection(int p, int dy, int dt, int dx, int nx, int ny, const float* theta,
                   const float* data, const float* recon, float* update, float* rot,
                   float* tmp, cuda::kernel_params& params, cuda::stream_t stream,
                   int eInterp = cuda::interpolation::nn())
{
    // calculate some values
    float theta_p_rad = fmodf(theta[p], twopi);
    int   block       = params.block;
    int   grid        = params.compute(nx, block);

    // reset destination arrays (NECESSARY! or will cause NaNs)
    // only do once bc for same theta, same pixels get overwritten
    cuda::memset(rot, 0, dy * nx * ny, stream);
    cuda::memset(tmp, 0, dy * nx * ny, stream);

    int smem = 0;
    for(int s = 0; s < dy; ++s)
    {
        const float* _recon  = recon + s * nx * ny;
        const float* _data   = data + s * dt * dx;
        float*       _update = update + s * nx * ny;
        float*       _rot    = rot + s * nx * ny;
        float*       _tmp    = tmp + s * nx * ny;

        // forward-rotate
        cuda::rotate(_rot, _recon, -theta_p_rad, nx, ny, stream, eInterp);

        // compute simdata
        pixels_kernel<<<grid, block, smem, stream>>>(p, nx, dx, _rot, _data);
        CUDA_CHECK_LAST_ERROR(stream);

        // back-rotate
        cuda::rotate(_tmp, _rot, theta_p_rad, nx, ny, stream, eInterp);

        // update shared update array
        cuda::atomic_sum<<<grid, block, 0, stream>>>(_update, _tmp, nx * ny);
        CUDA_CHECK_LAST_ERROR(stream);
    }
}
}

//======================================================================================//

void
sirt_cuda(const float* cpu_data, int dy, int dt, int dx, const float*, const float* theta,
          float* cpu_recon, int nx, int ny, int num_iter, RuntimeOptions* opts)
{
    printf("[%lu]> %s : nitr = %i, dy = %i, dt = %i, dx = %i, nx = %i, ny = %i\n",
           this_thread_id(), __FUNCTION__, num_iter, dy, dt, dx, nx, ny);

    // thread counter for device assignment
    static std::atomic<int> ntid;

    // compute some properties (expected python threads, max threads, device assignment)
    int pythread_num = ntid++;
    int device       = pythread_num % cuda::device_count();  // assign to device

    TIMEMORY_AUTO_TIMER("");

    // GPU allocated copies
    cuda::set_device(device);
    printf("[%lu] Running on device %i...\n", this_thread_id(), device);

    uintmax_t recon_pixels = static_cast<uintmax_t>(dy * nx * ny);
    auto      params       = cuda::kernel_params(opts->block_size[0], opts->grid_size[0]);
    auto      streams      = cuda::stream_create(opts->pool_size);
    float*    update       = cuda::malloc<float>(recon_pixels);
    float*    tmp_rot      = cuda::malloc<float>(streams.size() * recon_pixels);
    float*    tmp_tmp      = cuda::malloc<float>(streams.size() * recon_pixels);
    float*    recon        = cuda::malloc<float>(recon_pixels);
    float*    data         = cuda::malloc<float>(dy * dt * dx);
    cuda::memcpy(recon, cpu_recon, recon_pixels, cuda::host_to_device_v, 0);
    cuda::memcpy(data, cpu_data, dy * dt * dx, cuda::host_to_device_v, 0);
    uint32_t* sum_dist = cuda::compute_sum_dist(dy, dt, dx, nx, ny, theta, params);

    auto block = params.block;
    auto grid  = params.compute(recon_pixels, block);
    auto smem  = 0 * recon_pixels * sizeof(float);

    TOMOPY_NVXT_RANGE_PUSH(&nvtx_total);

    for(int i = 0; i < num_iter; i++)
    {
        // timing and profiling
        TIMEMORY_AUTO_TIMER("");
        TOMOPY_NVXT_RANGE_PUSH(&nvtx_iteration);
        START_TIMER(t_start);

        // sync the main stream
        cuda::stream_sync(0);

        // reset global update and sum_dist
        cuda::memset(update, 0, recon_pixels, 0);

        // sync
        for(auto& itr : streams)
            cuda::stream_sync(itr);

        // execute the loop over slices and projection angles
        // loop over independent projection angles
        for(int p = 0; p < dt; ++p)
        {
            auto offset = p % streams.size();
            // calculate offset for the streams
            float* off_rot = tmp_rot + (offset * recon_pixels);
            float* off_tmp = tmp_tmp + (offset * recon_pixels);
            compute_projection(p, dy, dt, dx, nx, ny, theta, data, recon, update, off_rot,
                               off_tmp, params, streams.at(offset));
        }

        // sync the thread streams
        for(auto& itr : streams)
            cuda::stream_sync(itr);

        // sync the main stream
        cuda::stream_sync(0);

        // update the global recon with global update and sum_dist
        update_kernel<<<grid, block, smem>>>(recon, update, sum_dist, dx, recon_pixels);

        // stop profile range and report timing
        TOMOPY_NVXT_RANGE_POP(0);
        REPORT_TIMER(t_start, "iteration", i, num_iter);
    }

    // copy to cpu
    cuda::memcpy(cpu_recon, recon, recon_pixels, cuda::device_to_host_v, 0);

    // sync and destroy main stream
    cuda::stream_destroy(streams);

    // cleanup
    cuda::free(recon);
    cuda::free(data);
    cuda::free(update);
    cuda::free(sum_dist);

    TOMOPY_NVXT_RANGE_POP(0);

    // sync the device
    cuda::device_sync();
}

//======================================================================================//
